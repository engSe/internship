//#include <fstream>
//#include <iostream>
//#include <iterator>
//#include <vector>
#include <string.h>
//#include <type_traits>
//#include <sstream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hipsolver.h>
#include <hipsparse.h>
#include <time.h>
#include <stdio.h>



//using namespace std;

template <typename T>
T* vector_insert(int n, char* filename, T a);
void vector_output(int n, double * vectors, char* filename);
void solve(int nnz, int  n, double tol, double* dVal, int * dCol, int * dRow, double* dbvec, double* dx);

int main(){

	int n, nnz;
	// initial val 
	//	// << "n and nnz" << endl;

	scanf("%d %d", &n, &nnz);

	//csr 
	double *result = new double[n];
	int *rowPtr = new   int[n + 1];
	int *colidx = new   int[nnz];
	double *csrval = new double[nnz];
	double *bvec = new double[n];


	// unsign int range
	if (nnz> 4294967295)
		return 1;

	if (n > 4294967295)
		printf("warn");

	//input :::::::::input은 ascii code로 저장되어야 한다.



	char * finb = "sysb.mat";
	char* finROW = "rowPtr.mat";
	char* finCOL = "colidx.mat";
	char* finVAL = "val.mat";

	double dou = 1.0;
	int uint = 1;

	bvec = vector_insert(n, finb, dou);
	rowPtr = vector_insert(n + 1, finROW, uint);
	colidx = vector_insert(nnz, finCOL, uint);
	csrval = vector_insert(nnz, finVAL, dou);



	//cuda alloc

	int* dCol, *dRow;
	double* dVal, *dbvec, *dx;
	hipError_t error;

	hipMalloc((void**)&dx, sizeof(double)*n);
	hipMalloc((void**)&dbvec, sizeof(double)*n);
	hipMalloc((void**)&dCol, sizeof(int)*nnz);
	hipMalloc((void**)&dRow, sizeof(int)*(n + 1));
	hipMalloc((void**)&dVal, sizeof(double)*nnz);



	hipMemcpy(dbvec, bvec, sizeof(double)*n, hipMemcpyHostToDevice);
	hipMemcpy(dCol, colidx, sizeof(int)*nnz, hipMemcpyHostToDevice);
	hipMemcpy(dRow, rowPtr, sizeof(int)*(n + 1), hipMemcpyHostToDevice);
	hipMemcpy(dVal, csrval, sizeof(double)*nnz, hipMemcpyHostToDevice);

	error = hipGetLastError();
	// << "Error status after hipMemcpy in getmemInfo: " << error << std::endl;

	//create and initialize library handles
	hipsolverSpHandle_t cusolver_handle;
	hipsparseHandle_t cusparse_handle;
	hipsolverStatus_t cusolver_status;
	hipsparseStatus_t cusparse_status;
	cusparse_status = hipsparseCreate(&cusparse_handle);
	// << "status hipsparseCreate: " << cusparse_status << std::endl;
	cusolver_status = hipsolverSpCreate(&cusolver_handle);
	// << "status hipsolverSpCreate: " << cusolver_status << std::endl;
	// solve
	hipDeviceSynchronize();

	double tol = 1e-6;
	// --- prepare solving and copy to GPU:
	int reorder = 0;
	int singularity = 0;

	// create matrix descriptor
	hipsparseMatDescr_t descrA;
	cusparse_status = hipsparseCreateMatDescr(&descrA);
	// << "status cusparse createMatDescr: " << cusparse_status << std::endl;

	hipDeviceSynchronize();

	//solve the system
	cusolver_status = hipsolverSpDcsrlsvqr(cusolver_handle, n, nnz, descrA, dVal,
		dRow, dCol, dbvec, tol, reorder, dx,
		&singularity);

	hipDeviceSynchronize();

	error = hipGetLastError();
	// << "Error status after solve(): " << error << std::endl;

	hipDeviceSynchronize();



	// return


	hipMemcpy(result, dx, n*sizeof(double), hipMemcpyDeviceToHost);


	// OUTPUT

	vector_output(n, result, "x_val.txt");

	//free

	hipFree(dCol);
	hipFree(dRow);
	hipFree(dx);
	hipFree(dbvec);
	hipFree(dVal);


	return 0;
}


//cuda function



/// 입출력
void vector_output(int n, double * vectors, char* filename){

	FILE *file;
	fopen_s(&file, filename, "w");
	int i = 0;
	while (i<n)
	{
		fprintf_s(file, "%f", vectors[i]);
		i++;
	}
	fclose(file);
}
template <typename T>
T* vector_insert(int n, char* filename, T a){


	FILE * file;
	fopen_s(&file, filename, "r");

	if (!file)
	{
		// << "file input error" << endl;

	}

	T *vectors = new T[n];
	int i = 0;
	while (i<n &&file){
		fscanf_s(file, "%f", vectors[i], sizeof(T));

		i++;

	}

	fclose(file);
	return vectors;
}
