#include <fstream>
#include <iostream>
#include <iterator>
#include <vector>
#include <string>
#include <type_traits>
#include <sstream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hipsolver.h>
#include <hipsparse.h>


double* vector_insert(int n,ifstream file);
void vector_output(double * vectors,ofstream file);


using namespace std;

int main(){

// initial val 
cout<<"n and nnz"<<endl;

cin<<n<<nnz;

            //csr 
double *result=new double[n];
unsigned int *rowPtr = new unsigned int[n+1];
unsigned int *colidx = new unsigned int[nnz];
double *csrval = new double[nnz]; 
double *bvec =new double[n];


// unsign int range
if (nnz> 4294967295)
return 1;

if(n>4294967295)
cout<<"warn"<<endl;

//input

ifstream finb,finROW,finCOL,finVAL;

finb.open("sysb.txt");
finROW.open("rowPtr.txt");
finCOL.open("col.txt");
finVAL.open("val.txt");

if(!finb||!finROW||!finCOL||!finVAL)
{
  cout<<"file input error"<<endl;
  return 1;
}


bvec=vector_insert(n,finb);
rowPtr=vector_insert(n+1,finROW);
colidx=vector_insert(nnz,finCOL);
csrval=vector_insert(nnz,finVAL);



finb.close();
finCOL.close();
finROW.close();
finVAL.close();


//cuda alloc

  unsigned int* dCol, *dRow;
	double* dVal;
  hipError_t error;

  hipMalloc((void**)&dCol, sizeof(int)*nnz);
	hipMalloc((void**)&dRow, sizeof(int)*(n + 1));
	hipMalloc((void**)&dVal, sizeof(double)*nnz);
	
	hipMemcpy(dCol, colidx, sizeof(int)*nnz, hipMemcpyHostToDevice);
	hipMemcpy(dRow, rowPtr, sizeof(int)*(n + 1), hipMemcpyHostToDevice);
	hipMemcpy(dVal, csrval, sizeof(double)*nnz, hipMemcpyHostToDevice);



// solve





return 0;
}


//cuda function

void solve(){
  
  
  
}

/// 입출력

template <typename T>
T* vector_insert(int n,ifstream file){

T *vectors=new T[n];
int i=0;
while(i<n &&!file.eof()   ){
file<<vectors[i];

i++;

}

return vectors;
}
