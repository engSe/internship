#include <fstream>
#include <iostream>
#include <iterator>
#include <vector>
#include <string>
#include <type_traits>
#include <sstream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hipsolver.h>
#include <hipsparse.h>
#include <time.h>

template <typename> vector_insert(int n,ifstream file,T a);
void vector_output(double * vectors,string filename);


using namespace std;

int main(){

// initial val 
cout<<"n and nnz"<<endl;

cin<<n<<nnz;

            //csr 
double *result=new double[n];
unsigned int *rowPtr = new unsigned int[n+1];
unsigned int *colidx = new unsigned int[nnz];
double *csrval = new double[nnz]; 
double *bvec =new double[n];


// unsign int range
if (nnz> 4294967295)
return 1;

if(n>4294967295)
cout<<"warn"<<endl;

//input :::::::::input은 ascii code로 저장되어야 한다.

ifstream finb,finROW,finCOL,finVAL;

string finb="sysb.mat";
string finROW = "rowPtr.mat";
string finCOL = "colidx.mat";
string finVAL = "val.mat";

double dou=1.0;
unsigned int uint=1;

bvec=vector_insert(n,finb,dou);
rowPtr=vector_insert(n+1,finROW,uint);
colidx=vector_insert(nnz,finCOL,uint);
csrval=vector_insert(nnz,finVAL,dou);



//cuda alloc

  unsigned int* dCol, *dRow;
	double* dVal;
  hipError_t error;

	hipMalloc((void**)&dCol, sizeof(int)*nnz);
	hipMalloc((void**)&dRow, sizeof(int)*(n + 1));
	hipMalloc((void**)&dVal, sizeof(double)*nnz);
	
	hipMemcpy(dCol, colidx, sizeof(int)*nnz, hipMemcpyHostToDevice);
	hipMemcpy(dRow, rowPtr, sizeof(int)*(n + 1), hipMemcpyHostToDevice);
	hipMemcpy(dVal, csrval, sizeof(double)*nnz, hipMemcpyHostToDevice);

	error = hipGetLastError();
	std::cout << "Error status after hipMemcpy in getmemInfo: " << error << std::endl;

	//create and initialize library handles
	hipsolverSpHandle_t cusolver_handle;
	hipsparseHandle_t cusparse_handle;
	hipsolverStatus_t cusolver_status;
	hipsparseStatus_t cusparse_status;
	cusparse_status = hipsparseCreate(&cusparse_handle);
	std::cout << "status hipsparseCreate: " << cusparse_status << std::endl;
	cusolver_status = hipsolverSpCreate(&cusolver_handle);
	std::cout << "status hipsolverSpCreate: " << cusolver_status << std::endl;
// solve





return 0;
}


//cuda function

void solve(){
  
  
  
}

/// 입출력
void vector_output(double * vectors,string filename){
	
	ofstream file(filename);
	int i=0;
	while(vectors)
	{
		file<<vectors;
		vectors++;
	}
}
template <typename T>
T* vector_insert(int n,string filename,T a){


ifstream file;
file.open(filename);

if(!file)
{
  cout<<"file input error"<<endl;
  return 1;
}

T *vectors=new T[n];
int i=0;
while(i<n &&file   ){
file<<vectors[i];

i++;

}

fclose(file);
return vectors;
}
