#include <fstream>
#include <iostream>
#include <iterator>
#include <vector>
#include <string>
#include <type_traits>
#include <sstream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hipsolver.h>
#include <hipsparse.h>
#include <time.h>




using namespace std;

template <typename T> 
T* vector_insert(int n, string filename, T a);
void vector_output(int n, double * vectors, string filename);
void solve(int nnz, int  n, double tol, double* dVal,   int * dCol,   int * dRow, double* dbvec, double* dx);

int main(){

	int n, nnz;
	// initial val 
	cout << "n and nnz" << endl;

	cin >> n >> nnz;

	//csr 
	double *result = new double[n];
	  int *rowPtr = new   int[n + 1];
	  int *colidx = new   int[nnz];
	double *csrval = new double[nnz];
	double *bvec = new double[n];


	// unsign int range
	if (nnz> 4294967295)
		return 1;

	if (n>4294967295)
		cout << "warn" << endl;

	//input :::::::::input은 ascii code로 저장되어야 한다.



	string finb = "sysb.mat";
	string finROW = "rowPtr.mat";
	string finCOL = "colidx.mat";
	string finVAL = "val.mat";

	double dou = 1.0;
	  int uint = 1;

	bvec = vector_insert(n, finb, dou);
	rowPtr = vector_insert(n + 1, finROW, uint);
	colidx = vector_insert(nnz, finCOL, uint);
	csrval = vector_insert(nnz, finVAL, dou);



	//cuda alloc

	  int* dCol, *dRow;
	double* dVal, *dbvec, *dx;
	hipError_t error;

	hipMalloc((void**)&dx, sizeof(double)*n);
	hipMalloc((void**)&dbvec, sizeof(double)*n);
	hipMalloc((void**)&dCol, sizeof(int)*nnz);
	hipMalloc((void**)&dRow, sizeof(int)*(n + 1));
	hipMalloc((void**)&dVal, sizeof(double)*nnz);



	hipMemcpy(dbvec, bvec, sizeof(double)*n, hipMemcpyHostToDevice);
	hipMemcpy(dCol, colidx, sizeof(int)*nnz, hipMemcpyHostToDevice);
	hipMemcpy(dRow, rowPtr, sizeof(int)*(n + 1), hipMemcpyHostToDevice);
	hipMemcpy(dVal, csrval, sizeof(double)*nnz, hipMemcpyHostToDevice);

	error = hipGetLastError();
	std::cout << "Error status after hipMemcpy in getmemInfo: " << error << std::endl;

	//create and initialize library handles
	hipsolverSpHandle_t cusolver_handle;
	hipsparseHandle_t cusparse_handle;
	hipsolverStatus_t cusolver_status;
	hipsparseStatus_t cusparse_status;
	cusparse_status = hipsparseCreate(&cusparse_handle);
	std::cout << "status hipsparseCreate: " << cusparse_status << std::endl;
	cusolver_status = hipsolverSpCreate(&cusolver_handle);
	std::cout << "status hipsolverSpCreate: " << cusolver_status << std::endl;
	// solve
	hipDeviceSynchronize();

	double tol = 1e-6;
	solve(nnz, n, tol, dVal, dCol, dRow, dbvec, dx);

	error = hipGetLastError();
	std::cout << "Error status after solve(): " << error << std::endl;

	hipDeviceSynchronize();



	// return


	hipMemcpy(result, dx, n*sizeof(double), hipMemcpyDeviceToHost);


	// OUTPUT

	vector_output(n, result, "x_val.txt");

	//free

	hipFree(dCol);
	hipFree(dRow);
	hipFree(dx);
	hipFree(dbvec);
	hipFree(dVal);


	return 0;
}


//cuda function

void solve(int nnz, int  n, double tol, double* dVal,   int* dCol,   int * dRow, double* dbvec, double* dx){


	// --- prepare solving and copy to GPU:
	int reorder = 0;
	int singularity = 0;

	// create matrix descriptor
	hipsparseMatDescr_t descrA;
	cusparse_status = hipsparseCreateMatDescr(&descrA);
	std::cout << "status cusparse createMatDescr: " << cusparse_status << std::endl;

	hipDeviceSynchronize();

	//solve the system
	cusolver_status = hipsolverSpDcsrlsvqr(cusolver_handle, m, nnz, descrA, dcsrVal,
		dcsrRowPtr, dcsrColInd, db, tol, reorder, dx,
		&singularity);

	hipDeviceSynchronize();
}

/// 입출력
void vector_output(int n, double * vectors, string filename){

	ofstream file(filename);
	int i = 0;
	while (i<n)
	{
		file << vectors[i];
		i++;
	}
	file.close();
}
template <typename T>
T* vector_insert(int n, string filename, T a){


	ifstream file;
	file.open(filename);

	if (!file)
	{
		cout << "file input error" << endl;
		return 1;
	}

	T *vectors = new T[n];
	int i = 0;
	while (i<n &&file){
		file << vectors[i];

		i++;

	}

	file.close();
	return vectors;
}
