#include <fstream>
#include <iostream>

#include <string.h>

#include <hip/hip_runtime.h>

#include <hipsolver.h>
#include <hipsparse.h>
#include <time.h>
#include <stdio.h>



using namespace std;

template <typename T>
T* vector_insert(int n, string filename, T a);

void vector_output(int n, double * vectors, string filename);

int main(){

	int n, nnz;
	// initial val 
	cout << "n and nnz" << endl;

	cin >> n >> nnz;

	clock_t init=clock();
	//csr 
	double *result = new double[n];
	int *rowPtr = new   int[n + 1];
	int *colidx = new   int[nnz];
	double *csrval = new double[nnz];
	double *bvec = new double[n];


	// unsign int range
	if (nnz > 4294967295)
	{
		cout << 'e';
		return 1;
	}
	if (n > 4294967295)
		printf("warn");

	//input :::::::::input은 ascii code로 저장되어야 한다.

	clock_t start=clock();
	cout << 't' << start - init << endl;

	/*string finb = "sysb.mat";
	string finROW = "rowPtr.mat";
	string finCOL = "colidx.mat";
	string finVAL = "val.mat";
*/

	string finb = "ex_b.mat";
	string finROW = "ex_row.mat";
	string finCOL = "ex_col.mat";
	string finVAL = "ex_val.mat";
	string outtxt = "ex_x_val.txt";
	double dou = 1.0;
	int uint = 1;

	


	bvec = vector_insert(n, finb, dou);
	rowPtr = vector_insert(n + 1, finROW, uint);

	colidx = vector_insert(nnz, finCOL, uint);
	csrval = vector_insert(nnz, finVAL, dou);


	clock_t insert=clock();

	cout << "insert t : " << insert - init<<endl;
	//cuda alloc

	int* dCol, *dRow;
	double* dVal, *dbvec, *dx;
	hipError_t error;

	hipMalloc((void**)&dx, sizeof(double)*n);
	hipMalloc((void**)&dbvec, sizeof(double)*n);
	hipMalloc((void**)&dCol, sizeof(int)*nnz);
	hipMalloc((void**)&dRow, sizeof(int)*(n + 1));
	hipMalloc((void**)&dVal, sizeof(double)*nnz);



	hipMemcpy(dbvec, bvec, sizeof(double)*n, hipMemcpyHostToDevice);
	hipMemcpy(dCol, colidx, sizeof(int)*nnz, hipMemcpyHostToDevice);
	hipMemcpy(dRow, rowPtr, sizeof(int)*(n + 1), hipMemcpyHostToDevice);
	hipMemcpy(dVal, csrval, sizeof(double)*nnz, hipMemcpyHostToDevice);

	error = hipGetLastError();
	cout << "Error status after hipMemcpy in getmemInfo: " << error << std::endl;


	
	clock_t cudamem=clock();
	cout << "cuda mem t : " << cudamem - insert << endl;

	//create and initialize library handles
	hipsolverSpHandle_t cusolver_handle;
	hipsparseHandle_t cusparse_handle;
	hipsolverStatus_t cusolver_status;
	hipsparseStatus_t cusparse_status;
	cusparse_status = hipsparseCreate(&cusparse_handle);
	cout << "status cusparseCreate: " << cusparse_status << std::endl;
	cusolver_status = hipsolverSpCreate(&cusolver_handle);
	cout << "status cusolverSpCreate: " << cusolver_status << std::endl;
	// solve
	hipDeviceSynchronize();

	double tol = 1e-6;
	// --- prepare solving and copy to GPU:
	int reorder = 0;
	int singularity = 0;

	// create matrix descriptor
	hipsparseMatDescr_t descrA;
	cusparse_status = hipsparseCreateMatDescr(&descrA);
	cout << "status cusparse createMatDescr: " << cusparse_status << std::endl;

	hipDeviceSynchronize();

	clock_t culib=clock();
	cout << "cuda lib t : " << culib - cudamem<<endl;

	//solve the system
	cusolver_status = hipsolverSpDcsrlsvqr(cusolver_handle, n, nnz, descrA, dVal,
		dRow, dCol, dbvec, tol, reorder, dx,
		&singularity);

	hipDeviceSynchronize();

	error = hipGetLastError();
	cout << "Error status after solve(): " << error << std::endl;

	hipDeviceSynchronize();

	clock_t solv=clock();

	cout << "solve t : "<< solv - culib << endl;
	// return


	hipMemcpy(result, dx, n*sizeof(double), hipMemcpyDeviceToHost);
	cout << "total cuda t : " << clock() - insert << endl;

	// OUTPUT

	vector_output(n, result, outtxt);
	vector_output(n, (double*)rowPtr, "ex_row.txt");

	clock_t output=clock();
	cout << "out t : " << output - solv << endl;


	//free mem

	hipFree(dCol);
	hipFree(dRow);
	hipFree(dx);
	hipFree(dbvec);
	hipFree(dVal);


	return 0;
}


//cuda function



/// 입출력
void vector_output(int n, double * vectors, string filename){

	ofstream file(filename);
	int i = 0;
	while (i<n)
	{
		file << vectors[i]<<'\n';
		i++;
	}
	file.close();
}
template <typename T>
T* vector_insert(int n, string filename, T a){

	ifstream file;
	file.open( filename);

	if (!file)
	{
		cout << "file input error" << endl;

	}

	T *vectors = new T[n];
	int i = 0;
	while (i<n &&file){
		file >> vectors[i];
		i++;
	}

	file.close();
	return vectors;
}
